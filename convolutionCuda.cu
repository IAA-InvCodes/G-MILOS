#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>
#include "definesCuda.cuh"
#include "defines.h"
#include "convolutionCuda.cuh"



////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex scale
__device__ __host__ hipfftDoubleComplex ComplexScale(hipfftDoubleComplex  a, float s)
{
    hipfftDoubleComplex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

// Complex multiplication
__device__ __host__ hipfftDoubleComplex ComplexMul(hipfftDoubleComplex a, hipfftDoubleComplex b)
{
    hipfftDoubleComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

// Complex pointwise multiplication
/**
* @param a: Array with operand a of multiplication 
* @param b: array with operand b of multpilication 
* @param c: array with result of multiplication 
* @param size: length of param arrays 
* @param scale: scale factor for multiplication 
*/
__global__ void ComplexPointwiseMulAndScale(const hipfftDoubleComplex *a, const hipfftDoubleComplex *b, hipfftDoubleComplex * c, int size, float scale)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < size; i += numThreads)
    {
        c[i] = ComplexMul(a[i], ComplexScale(b[i], scale));
    }
}

/**
* 
**/
__global__ void Scale_PSF_FFT(hipfftDoubleComplex *a, int size, float scale)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = ComplexScale(a[i], scale);
    }
}